#include "hip/hip_runtime.h"
#include "Algorithm.hpp"
#include <numeric>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

const std::size_t TOTAL_POINTS = 1e9;
const std::size_t THREADS_NUM = 32;
const std::size_t BLOCKS_NUM = 640;
const std::size_t THREAD_ITERATIONS = TOTAL_POINTS / THREADS_NUM / BLOCKS_NUM;


__global__ void monteCarlo_cuda(std::size_t* totals)
{
	__shared__ std::size_t counter[THREADS_NUM];

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	counter[threadIdx.x] = 0;

	hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

	for (int i = 0; i < THREAD_ITERATIONS; i++)
	{
		float x = hiprand_uniform(&rng);
		float y = hiprand_uniform(&rng);
		counter[threadIdx.x] += 1 - int(x * x + y * y);
	}

	__syncthreads();

	if (threadIdx.x == 0)
	{
		totals[blockIdx.x] = 0;
		for (int i = 0; i < THREADS_NUM; i++)
		{
			totals[blockIdx.x] += counter[i];
		}
	}
}

MonteCarloResult monteCarlo()
{
	thrust::host_vector<std::size_t> blocksCount(BLOCKS_NUM);
	thrust::device_vector<std::size_t> blocksCount_dev(BLOCKS_NUM);

	monteCarlo_cuda << <BLOCKS_NUM, THREADS_NUM >> > (blocksCount_dev.data().get());
	thrust::copy(blocksCount_dev.begin(), blocksCount_dev.end(), blocksCount.begin());

	const auto pointsInCircle = std::accumulate(blocksCount.cbegin(), blocksCount.cend(), std::size_t{});
	return { pointsInCircle, TOTAL_POINTS };
}